#include "hip/hip_runtime.h"
#include <functional>
#include <vector>
#include <map>
#include <memory>
#include <cstdint>
#include <cstddef>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <sisci_types.h>
#include <sisci_api.h>
#include "rpc.h"
#include "segment.h"
#include "transfer.h"
#include "benchmark.h"
#include "util.h"
#include "log.h"
#include "args.h"


typedef std::map<uint, BufferPtr> BufferMap;
typedef std::map<uint, int> DeviceMap;


/* Fill buffers with a random value */
static void fillBuffers(const SegmentMap& segments, const BufferMap& buffers, const DeviceMap& devices)
{
    for (auto segmentIt = segments.begin(); segmentIt != segments.end(); ++segmentIt)
    {
        const SegmentPtr segment = segmentIt->second;

        srand(currentTime());
        const uint8_t randomValue = (rand() & 0xee) | 2; // should never be 0x00 or 0xff

        auto bufferIt = buffers.find(segment->id);
        if (bufferIt != buffers.end())
        {
            Log::info("Filling segment %u with value %02X, this might take some time...",
                    segment->id, randomValue);

            fillBuffer(devices.at(segment->id), bufferIt->second, segment->size, randomValue);
        }
        else
        {
            Log::info("Filling RAM segment %u with value %02X, this might take some time...",
                    segment->id, randomValue);

            fillSegment(segment->getSegment(), segment->size, randomValue);
        }
    }
}


/* Iterate over segment infos and create segments accordingly */
static void createSegments(SegmentSpecMap& segmentSpecs, SegmentMap& segments, BufferMap& buffers, DeviceMap& devices)
{
    for (auto segmentIt = segmentSpecs.begin(); segmentIt != segmentSpecs.end(); ++segmentIt)
    {
        SegmentSpecPtr& spec = segmentIt->second;
        SegmentPtr segment;

        if (spec->deviceId != NO_DEVICE)
        {
            BufferPtr buffer(allocDeviceMem(spec->deviceId, spec->size));
            buffers[spec->segmentId] = buffer;
            devices[spec->segmentId] = spec->deviceId;

            void* devicePtr = getDevicePtr(buffer);
            segment = Segment::createWithPhysMem(spec->segmentId, spec->size, spec->adapters, spec->deviceId, devicePtr, spec->flags);
        }
        else
        {
            segment = Segment::create(spec->segmentId, spec->size, spec->adapters, spec->flags);
        }

        segments[segment->id] = segment;
    }
}


/* Iterate over transfer infos and create transfers */
static void createTransfers(const DmaJobList& jobSpecs, TransferList& transfers, const SegmentMap& segments)
{
    for (const auto job: jobSpecs)
    {
        // Find corresponding local segment
        auto segment = segments.find(job->localSegmentId);
        if (segment == segments.end())
        {
            Log::error("Could not match local segment %u", job->localSegmentId);
            throw std::string("Could not find local segment ") + std::to_string(job->localSegmentId);
        }

        const SegmentPtr& localSegment = segment->second;

        // Notify user about potential error condition with combination of segment and transfer flags
        switch ((!!(localSegment->flags & SCI_FLAG_DMA_GLOBAL) << 1) | !!(job->flags & SCI_FLAG_DMA_GLOBAL))
        {
            case 2:
                Log::info("Segment %u is created with SCI_FLAG_DMA_GLOBAL but transfer is not", localSegment->id);
                break;

            case 1:
                Log::warn("Transfer specifies SCI_FLAG_DMA_GLOBAL but local segment %u is not", localSegment->id);
                break;

            case 0:
            case 3:
                break;
        }

        if (!!(job->flags & SCI_FLAG_DMA_GLOBAL) && !!(job->flags & SCI_FLAG_DMA_SYSDMA))
        {
            Log::warn("Both SCI_FLAG_DMA and SCI_FLAG_DMA_SYSDMA are set");
        }

        // Check if segment is prepared on adapter
        if (localSegment->adapters.find(job->localAdapterNo) == localSegment->adapters.end())
        {
            Log::warn("Segment %u is not prepared on adapter %u", localSegment->id, job->localAdapterNo);
        }

        // Check transfer vector size
        if (job->vector.size() > DIS_DMA_MAX_VECLEN)
        {
            Log::warn("DMA transfer vector exceeds %zu elements (DIS_DMA_MAX_VECLEN)", DIS_DMA_MAX_VECLEN);
        }

        // Connect to remote end and create transfer handle
        TransferPtr transfer = Transfer::create(localSegment, job->remoteNodeId, job->remoteSegmentId, job->localAdapterNo, job->flags);

        const size_t remoteSegmentSize = transfer->remoteSegmentSize;

        // Add transfer vector entries
        for (const dis_dma_vec_t& vecEntry: job->vector)
        {
            if (vecEntry.local_offset + vecEntry.size > localSegment->size)
            {
                Log::error("Transfer size exceeds size of local segment %u", localSegment->id);
                throw std::string("Transfer size exceeds size of local segment ") + std::to_string(localSegment->id);
            }
            else if (vecEntry.remote_offset + vecEntry.size > remoteSegmentSize)
            {
                Log::error("Transfer size exceeds size of remote segment %u on node %u", job->remoteSegmentId, job->remoteNodeId);
                throw std::string("Transfer size exceeds size of remote segment ") 
                    + std::to_string(job->remoteSegmentId) + " on node " + std::to_string(job->remoteNodeId);
            }

            transfer->addVectorEntry(vecEntry);
        }

        transfers.push_back(transfer);
    }
}


int main(int argc, char** argv)
{
    SegmentSpecMap segmentSpecs;
    DmaJobList transferSpecs;
    bool verify = false;

    // Parse command line arguments
    try
    {
        Log::Level logLevel = Log::Level::WARN;
        parseArguments(argc, argv, segmentSpecs, transferSpecs, logLevel, verify);
        Log::init(stderr, logLevel);
    }
    catch (int error)
    {
        return error;
    }
    catch (const std::string& error)
    {
        fprintf(stderr, "%s\n", error.c_str());
        return 1;
    }

    // Initialize SISCI API
    sci_error_t sciError;
    SCIInitialize(0, &sciError);
    if (sciError != SCI_ERR_OK)
    {
        Log::abort("Failed to initialize SISCI API");
        return 2;
    }

    SegmentMap segments;
    BufferMap buffers;
    DeviceMap devices;

    // Allocate buffers and create segments
    try
    {
        createSegments(segmentSpecs, segments, buffers, devices);
        fillBuffers(segments, buffers, devices);
    }
    catch (const std::string& error)
    {
        Log::abort("Failed to create segments: %s", error.c_str());
        return 1;
    }
    catch (const std::runtime_error& error)
    {
        Log::abort("Failed to create segments: %s", error.what());
        return 2;
    }

    // Create transfers
    TransferList transfers;
    try
    {
        createTransfers(transferSpecs, transfers, segments);
    }
    catch (const std::string& error)
    {
        Log::abort("Failed to create transfers: %s", error.c_str());
        return 1;
    }
    catch (const std::runtime_error& error)
    {
        Log::abort("Failed to create transfers: %s", error.what());
        return 2;
    }

    // Create checksum callback
    ChecksumCallback calc = [&buffers, &devices](const Segment& segment, size_t offset, size_t size, uint32_t& checksum) -> bool
    {
        Log::info("Calculating checksum for segment %u", segment.id);



        return false;
    };

    if (transfers.empty())
    {

        // No transfers specified, run as server
        if (runBenchmarkServer(segments, calc) != 0)
        {
            fprintf(stderr, "Server failed!\n");
        }
    }
    else if (verify)
    {
        // Validate transfers
        if (validateTransfers(transfers, calc, stdout) != 0)
        {
            fprintf(stderr, "Validation failed!\n");
        }
    }
    else
    {
        // Run benchmark client
        runBenchmarkClient(transfers, stdout);
    }

    // Nuke any active SISCI handles
    transfers.clear();
    segments.clear();

    // Free any GPU buffers
    buffers.clear();

    // Terminate SISCI API
    SCITerminate();

    return 0;
}


/* Print a list of local GPUs */
void listGpus()
{
    hipError_t err;

    // Get number of devices
    int deviceCount = 0;
    err = hipGetDeviceCount(&deviceCount);
    if (err != hipSuccess)
    {
        throw std::string(hipGetErrorString(err));
    }

    // Print header
    fprintf(stderr, "\n %2s   %-20s   %-9s   %8s   %7s   %7s   %8s   %6s   %3s   %15s\n",
            "ID", "Device name", "IO addr", "Comp mod", "Managed", "Unified", "Map hmem", "#Async", "L1", "Global mem size");
    fprintf(stderr, "------------------------------------------------------------------------------------------------------------------\n");

    // Iterate over devices and print properties
    for (int i = 0; i < deviceCount; ++i)
    {
        hipDeviceProp_t prop;

        err = hipGetDeviceProperties(&prop, i);
        if (err != hipSuccess)
        {
            throw std::string(hipGetErrorString(err));
        }

        fprintf(stderr, " %2d   %-20s   %02x:%02x.%-3x   %5d.%-2d   %7s   %7s   %8s   %6d   %3s   %10.02f MiB\n",
                i, prop.name, prop.pciBusID, prop.pciDeviceID, prop.pciDomainID,
                prop.major, prop.minor, 
                prop.managedMemory ? "yes" : "no", 
                prop.unifiedAddressing ? "yes" : "no",
                prop.canMapHostMemory ? "yes" : "no",
                prop.asyncEngineCount,
                prop.globalL1CacheSupported ? "yes" : "no",
                prop.totalGlobalMem / (double) (1 << 20)
               );
    }
    fprintf(stderr, "\n");
}

