#include "hip/hip_runtime.h"
#include <vector>
#include <map>
#include <exception>
#include <stdexcept>
#include <string>
#include <cstring>
#include <getopt.h>
#include <errno.h>
#include <sisci_types.h>
#include <sisci_api.h>
#include "task.h"
#include "util.h"

using std::vector;
using std::map;
using std::string;


static const char* logFilename = nullptr;
static uint logLevel = 0;


static struct option options[] = 
{
    { .name = "segment", .has_arg = true, .flag = nullptr, .val = 's' },
    { .name = "transfer", .has_arg = true, .flag = nullptr, .val = 't' },
    { .name = "verbose", .has_arg = false, .flag = nullptr, .val = 'v' },
    { .name = "verbosity", .has_arg = true, .flag = nullptr, .val = 'v' },
    { .name = "report", .has_arg = true, .flag = nullptr, .val = 'f' },
    { .name = "log", .has_arg = true, .flag = nullptr, .val = 'g' },
    { .name = "list", .has_arg = false, .flag = nullptr, .val = 'l' },
    { .name = "help", .has_arg = false, .flag = nullptr, .val = 'h' },
    { .name = nullptr, .has_arg = false, .flag = nullptr, .val = 0 }
};


/* Show program usage text */
static void giveUsage(const char* programName)
{
    fprintf(stderr, 
            "Usage: %s --segment <segment string>...\n"
            "   or: %s --segment <segment string>... --transfer <transfer string>...\n"
            "\nDescription\n"
            "    Benchmark the performance of GPU to GPU RDMA transfer.\n"
            "\nServer arguments\n"
            "    --segment  <segment string>    create a local segment\n"
            "\nClient arguments\n"
            "    --segment  <segment string>    create a local segment\n"
            "    --transfer <transfer string>   DMA transfer specification\n"
            "\nString format\n"
            "        key1=value1,key2,key3,key4=value4,key5=value5...\n"
            "\nSegment string\n"
            "    size=<size>                    specify size of the segment (required)\n"
            "    ls=<id>                        local segment id [default is 0]\n"
            "    a=<no>                         local adapter for segment [default is 0]\n"
            "    gpu=<gpu>                      specify local GPU to host buffer on [omit to host buffer in RAM]\n"
            "\nTransfer string\n"
            "    rn=<id>                        remote node id (required)\n"
            "    rs=<id>                        remote segment id [default is 0]\n"
            "    a=<no>                         local adapter for segment [default is 0]\n"
            "    ls=<id>                        local segment id [default is 0]\n"
            "    pull                           read data from remote buffer instead of writing\n"
            "    ro=<offset>                    offset into remote segment [default is 0]\n"
            "    lo=<offset>                    offset into local segment [default is 0]\n"
            "    size=<size>                    transfer size [default is the size of segment]\n"
            "    repeat=<count>                 number of times to repeat transfer [default is 1]\n"
            "    verify                         run memcmp() instead of calculating checksum\n"
            "\nOther options\n"
            "  --verbosity      <level>         specify \"error\", \"warn\", \"info\" or \"debug\" log level\n"
            "  --log            <filename>      use a log file instead of stderr for logging\n"
            "  --report         <filename>      use a report file instead of stdout\n"
            "  --list                           show a list of local GPUs and quit\n"
            "  --help                           show this help and quit\n"
            "\n"
            , programName, programName);
}


static void listGpus()
{
    hipError_t err;

    int deviceCount = 0;
    err = hipGetDeviceCount(&deviceCount);
    if (err != hipSuccess)
    {
        throw string(hipGetErrorString(err));
    }


    fprintf(stderr, "\n %2s   %-20s   %-9s   %12s   %7s   %7s   %8s   %6s   %3s   %15s\n",
            "ID", "Device name", "IO addr", "Compute mode", "Managed", "Unified", "Map hmem", "#Async", "L1", "Global mem size");
    fprintf(stderr, "---------------------------------------------------------------------------------------------------------------------\n");

    for (int i = 0; i < deviceCount; ++i)
    {
        hipDeviceProp_t prop;

        err = hipGetDeviceProperties(&prop, i);
        if (err != hipSuccess)
        {
            throw string(hipGetErrorString(err));
        }

        fprintf(stderr, " %2d   %-20s   %02x:%02x.%-3x   %9d.%-2d   %7s   %7s   %8s   %6d   %3s   %10.02f MiB\n",
                i, prop.name, prop.pciBusID, prop.pciDeviceID, prop.pciDomainID,
                prop.major, prop.minor, 
                prop.managedMemory ? "yes" : "no", 
                prop.unifiedAddressing ? "yes" : "no",
                prop.canMapHostMemory ? "yes" : "no",
                prop.asyncEngineCount,
                prop.globalL1CacheSupported ? "yes" : "no",
                prop.totalGlobalMem / (double) (1 << 20)
               );
    }
    fprintf(stderr, "\n");
}


/* Helper function for retrieving key--value pairs from option string */
static const char* nextToken(const char* str, string& key, string& value)
{
    bool readValue = false;

    while (true)
    {
        switch (*str)
        {
            case '=':
                if (readValue)
                {
                    throw string("Invalid string syntax");
                }
                readValue = true;
                break;

            case ',':
                return str + 1;

            case '\0':
                return str;

            default:
                if (readValue)
                {
                    value += *str;
                }
                else
                {
                    key += *str;
                }
                break;
        }

        ++str;
    }
}


/* Helper function for reading a number from a string */
static size_t parseNumber(const string& key, const string& value)
{
    if (value.empty())
    {
        throw string("String key '") + key + string("' expects a numerical value but got nothing");
    }

    size_t idx;

    size_t v = std::stoul(value, &idx, 0);

    if (idx != value.size())
    {
        throw string("String key '") + key + string("' expects a numerical value but got ``") + value + "''";
    }

    return v;
}


static void parseSegmentString(const char* segmentString, map<uint, Segment>& segments)
{
    Segment segment;
    segment.adapterNo = 0;
    segment.segmentId = 0;
    segment.deviceId = NO_DEVICE;
    segment.size = 0;

    // Parse segment string
    while (*segmentString != '\0')
    {
        string key, value;
        segmentString = nextToken(segmentString, key, value);

        if (key == "size" || key == "s")
        {
            segment.size = parseNumber(key, value);
        }
        else if (key == "local-segment-id" || key == "local-segment" || key == "ls")
        {
            segment.segmentId = parseNumber(key, value);
        }
        else if (key == "adapter" || key == "adapt" || key == "a")
        {
            segment.adapterNo = parseNumber(key, value);
        }
        else if (key == "device" || key == "gpu")
        {
            segment.adapterNo = parseNumber(key, value);
        }
        else if (!key.empty())
        {
            throw string("Unknown string key: ``") + key + "''";
        }
    }

    // Some sanity checking
    if (segment.size == 0)
    {
        throw string("Local segment size must be specified");
    }

    // Check if segment is already specified
    map<uint, Segment>::iterator i = segments.lower_bound(segment.segmentId);
    if (i == segments.end() || segment.segmentId < i->first)
    {
        segments.insert(i, std::make_pair(segment.segmentId, segment));
    }
    else
    {
        throw string("Local segment ") + std::to_string(segment.segmentId) + " was already specified";
    }
}


static void parseTransferString(const char* transferString, vector<Transfer>& transfers)
{
    Transfer transfer;
    transfer.remoteNodeId = 0;
    transfer.remoteSegmentId = 0;
    transfer.localAdapterNo = 0;
    transfer.localSegmentId = 0;
    transfer.size = 0;
    transfer.localOffset = 0;
    transfer.remoteOffset = 0;
    transfer.repeat = 1;
    transfer.verify = false;
    transfer.pull = false;
    transfer.global = false;

    // Parse transfer string
    while (*transferString != '\0')
    {
        string key, value;
        transferString = nextToken(transferString, key, value);

        if (key == "remote-node-id" || key == "remote-node" || key == "rn")
        {
            transfer.remoteNodeId = parseNumber(key, value);   
        }
        else if (key == "remote-segment-id" || key == "remote-segment" || key == "rs")
        {
            transfer.remoteSegmentId = parseNumber(key, value);
        }
        else if (key == "adapter" || key == "adapt" || key == "a")
        {
            transfer.localAdapterNo = parseNumber(key, value);
        }
        else if (key == "local-segment-id" || key == "local-segment" || key == "ls")
        {
            transfer.localSegmentId = parseNumber(key, value);
        }
        else if (key == "pull" || key == "read")
        {
            transfer.pull = true;
        }
        else if (key == "remote-offset" || key == "ro")
        {
            transfer.remoteOffset = parseNumber(key, value);
        }
        else if (key == "local-offset" || key == "lo")
        {
            transfer.localOffset = parseNumber(key, value);
        }
        else if (key == "size" || key == "s")
        {
            transfer.size = parseNumber(key, value);
        }
        else if (key == "repeat" || key == "c" || key == "r" || key == "n")
        {
            transfer.repeat = parseNumber(key, value);
        }
        else if (key == "verify")
        {
            transfer.verify = true;
        }
    }


    // Some sanity checking
    if (transfer.remoteNodeId == 0)
    {
        throw string("Remote node id must be specified for transfers");
    }

    if (transfer.repeat == 0)
    {
        throw string("Transfers can not be repeated 0 times");
    }

    transfers.push_back(transfer);
}


static uint parseVerbosity(const char* argument, uint level)
{
    if (argument == nullptr)
    {
        // No argument given, increase verbosity level
        return level + 1;
    }
    else if (strcmp(argument, "error") == 0)
    {
        return 0;
    }
    else if (strcmp(argument, "warn") == 0)
    {
        return 1;
    }
    else if (strcmp(argument, "info") == 0)
    {
        return 2;
    }
    else if (strcmp(argument, "debug") == 0)
    {
        return 3;
    }

    // Try to parse verbosity level as a number
    char* ptr = nullptr;
    level = strtoul(optarg, &ptr, 10);

    if (ptr == nullptr || *ptr != '\0')
    {
        throw "Unknown log level: ``" + string(optarg) + "''";
    }

    return level;
}


/* Parse command line options */
static void parseArguments(int argc, char** argv, map<uint, Segment>& segments, vector<Transfer>& transfers)
{
    int option;
    int index;

    while ((option = getopt_long(argc, argv, "-:s:t:f:g:v::lh", options, &index)) != -1)
    {
        switch (option)
        {
            case ':': // Missing value for option
                fprintf(stderr, "Argument %s requires a value\n", argv[optind - 1]);
                giveUsage(argv[0]);
                throw 1;

            case '?': // Unknown option
                fprintf(stderr, "Unknown option: ``%s''\n", argv[optind - 1]);
                giveUsage(argv[0]);
                throw 1;

            case 'h': // Show help
                giveUsage(argv[0]);
                throw 0;

            case 'l': // List GPUs
                listGpus();
                throw 0;

            case 'v': // Increase verbosity level
                logLevel = parseVerbosity(optarg, logLevel);
                break;

            case 'f': // Set report file
                break;

            case 'g': // Set log file
                logFilename = optarg;
                break;

            case 's': // Parse local segment options
                parseSegmentString(optarg, segments);
                break;

            case 't': // Parse transfer string
                parseTransferString(optarg, transfers);
                break;
        }
    }
}


int main(int argc, char** argv)
{
    map<uint, Segment> segments;
    vector<Transfer> transfers;

    // Parse command line arguments
    try
    {
        parseArguments(argc, argv, segments, transfers);
    }
    catch (int error)
    {
        return error;
    }
    catch (const string& error)
    {
        fprintf(stderr, "%s\n", error.c_str());
        return 1;
    }

    // Initialize logging
    FILE* logFile = stderr;
    if (logFilename != nullptr)
    {
        if ((logFile = fopen(logFilename, "a")) == nullptr)
        {
            fprintf(stderr, "Failed to open log file: %s\n", strerror(errno));
            return 1;
        }

        initLog(logFile, logLevel);
    }

    // Initialize SISCI API
    sci_error_t err;
    SCIInitialize(0, &err);
    if (err != SCI_ERR_OK)
    {
        error("Failed to initialize SISCI: %s", scierrstr(err));
        fprintf(stderr, "FAIL\n");
        return 2;
    }

    // Create segments and connections
    try
    {
        for (auto& segment: segments)
        {
            fprintf(stdout, "%u %zu\n", segment.first, segment.second.size);
        }
    }
    catch (sci_error_t error)
    {
    }

    // Terminate SISCI API
    SCITerminate();
    fprintf(stderr, "OK\n");
    fclose(logFile);
    return 0;
}
