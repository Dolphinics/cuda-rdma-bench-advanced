#include "hip/hip_runtime.h"
#include <functional>
#include <vector>
#include <map>
#include <memory>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <sisci_types.h>
#include <sisci_api.h>
#include "util.h"
#include "segment.h"
#include "transfer.h"
#include "benchmark.h"
#include "log.h"
#include "args.h"

typedef std::shared_ptr<void> BufferPtr;
typedef std::map<uint, BufferPtr> BufferMap;


/* Iterate over segment infos and create segments accordingly */
static void createSegments(SegmentSpecMap& segmentSpecs, SegmentList& segments, BufferMap& buffers)
{
    for (auto segmentIt = segmentSpecs.begin(); segmentIt != segmentSpecs.end(); ++segmentIt)
    {
        SegmentSpec& spec = segmentIt->second;
        SegmentPtr segment;

        if (spec.deviceId != NO_DEVICE)
        {
            const int deviceId = spec.deviceId;

            hipError_t err = hipSetDevice(deviceId);
            if (err != hipSuccess)
            {
                Log::error("Failed to initialize GPU %d: %s", deviceId, hipGetErrorString(err));
                throw std::string(hipGetErrorString(err));
            }
                
            void* bufferPtr;
            err = hipMalloc(&bufferPtr, spec.size);
            if (err != hipSuccess)
            {
                Log::error("Failed to allocate buffer on GPU %d: %s", deviceId, hipGetErrorString(err));
                throw std::string(hipGetErrorString(err));
            }

            Log::debug("Allocated buffer on GPU %d (%p)", deviceId, bufferPtr);

            auto release = [deviceId](void* buffer) {
                Log::debug("Freeing GPU buffer on device %d (%p)", deviceId, buffer);
                hipFree(buffer);
            };

            buffers[spec.segmentId] = BufferPtr(bufferPtr, release);

            void* devicePtr = getDevicePtr(bufferPtr);
            segment = Segment::createWithPhysMem(spec.segmentId, spec.size, spec.adapters, spec.deviceId, devicePtr);
        }
        else
        {
            segment = Segment::create(spec.segmentId, spec.size, spec.adapters);
        }

        segments.push_back(segment);
    }
}


/* Iterate over transfer infos and create transfers */
static void createTransfers(const TransferSpecList& transferSpecs, TransferList& transfers, const SegmentList& segments)
{
    for (const auto info: transferSpecs)
    {
        // Find corresponding local segment
        SegmentPtr localSegment(nullptr);
        for (SegmentPtr segment: segments)
        {
            if (segment->id == info->localSegmentId)
            {
                localSegment = segment;
                break;
            }
        }

        if (localSegment.get() == nullptr)
        {
            Log::error("Could not match local segment %u", info->localSegmentId);
            throw std::string("Could not find local segment ") + std::to_string(info->localSegmentId);
        }
        
        // Connect to remote end and create transfer
        TransferPtr transfer(new Transfer(localSegment, info->remoteNodeId, info->remoteSegmentId, info->localAdapterNo));
        for (const dis_dma_vec_t& vecEntry: info->vector)
        {
            transfer->addVectorEntry(vecEntry);
        }

        transfers.push_back(transfer);
    }
}


int main(int argc, char** argv)
{
    SegmentSpecMap segmentSpecs;
    TransferSpecList transferSpecs;

    // Parse command line arguments
    try
    {
        Log::Level logLevel = Log::Level::ERROR;
        parseArguments(argc, argv, segmentSpecs, transferSpecs, logLevel);
        Log::init(stderr, logLevel);
    }
    catch (int error)
    {
        return error;
    }
    catch (const std::string& error)
    {
        fprintf(stderr, "%s\n", error.c_str());
        return 1;
    }

    // Initialize SISCI API
    sci_error_t sciError;
    SCIInitialize(0, &sciError);
    if (sciError != SCI_ERR_OK)
    {
        Log::abort("Failed to initialize SISCI API");
        return 2;
    }

    // Allocate buffers and create segments
    SegmentList segments;
    BufferMap buffers;

    try
    {
        createSegments(segmentSpecs, segments, buffers);
    }
    catch (const std::string& error)
    {
        Log::abort("Failed to create segments: %s", error.c_str());
        return 1;
    }
    catch (const std::runtime_error& error)
    {
        Log::abort("Failed to create segments: %s", error.what());
        return 2;
    }

    // Create transfers
    TransferList transfers;
    try
    {
        createTransfers(transferSpecs, transfers, segments);
    }
    catch (const std::string& error)
    {
        Log::abort("Failed to create transfers: %s", error.c_str());
        return 1;
    }
    catch (const std::runtime_error& error)
    {
        Log::abort("Failed to create transfers: %s", error.what());
        return 2;
    }

    if (transfers.empty())
    {
        // Create local interrupt


        // No transfers specified, run as server
        if (runBenchmarkServer(segments) != 0)
        {
        }
    }
    else
    {
        // Run benchmark client
        if (runBenchmarkClient(segments, transfers) != 0)
        {
        }
    }

    // Nuke any active SISCI handles
    transfers.clear();
    segments.clear();

    // Free any GPU buffers
    buffers.clear();

    // Terminate SISCI API
    SCITerminate();

    return 0;
}


/* Print a list of local GPUs */
void listGpus()
{
    hipError_t err;

    // Get number of devices
    int deviceCount = 0;
    err = hipGetDeviceCount(&deviceCount);
    if (err != hipSuccess)
    {
        throw std::string(hipGetErrorString(err));
    }

    // Print header
    fprintf(stderr, "\n %2s   %-20s   %-9s   %8s   %7s   %7s   %8s   %6s   %3s   %15s\n",
            "ID", "Device name", "IO addr", "Comp mod", "Managed", "Unified", "Map hmem", "#Async", "L1", "Global mem size");
    fprintf(stderr, "-----------------------------------------------------------------------------------------------------------------\n");

    // Iterate over devices and print properties
    for (int i = 0; i < deviceCount; ++i)
    {
        hipDeviceProp_t prop;

        err = hipGetDeviceProperties(&prop, i);
        if (err != hipSuccess)
        {
            throw std::string(hipGetErrorString(err));
        }

        fprintf(stderr, " %2d   %-20s   %02x:%02x.%-3x   %5d.%-2d   %7s   %7s   %8s   %6d   %3s   %10.02f MiB\n",
                i, prop.name, prop.pciBusID, prop.pciDeviceID, prop.pciDomainID,
                prop.major, prop.minor, 
                prop.managedMemory ? "yes" : "no", 
                prop.unifiedAddressing ? "yes" : "no",
                prop.canMapHostMemory ? "yes" : "no",
                prop.asyncEngineCount,
                prop.globalL1CacheSupported ? "yes" : "no",
                prop.totalGlobalMem / (double) (1 << 20)
               );
    }
    fprintf(stderr, "\n");
}
