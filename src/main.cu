#include "hip/hip_runtime.h"
#include "segment.h"
#include "transfer.h"
#include <memory>
#include <vector>
#include <map>
#include <exception>
#include <stdexcept>
#include <string>
#include <cstring>
#include <getopt.h>
#include <errno.h>
#include <signal.h>
#include <sisci_api.h>
#include "util.h"
#include "log.h"
#include "server.h"
#include "client.h"

using std::string;


static struct option options[] = 
{
    { .name = "segment", .has_arg = true, .flag = nullptr, .val = 's' },
    { .name = "transfer", .has_arg = true, .flag = nullptr, .val = 't' },
    { .name = "verbosity", .has_arg = true, .flag = nullptr, .val = 'v' },
    { .name = "report", .has_arg = true, .flag = nullptr, .val = 'f' },
    { .name = "report-file", .has_arg = true, .flag = nullptr, .val = 'f' },
    { .name = "log", .has_arg = true, .flag = nullptr, .val = 'g' },
    { .name = "logfile", .has_arg = true, .flag = nullptr, .val = 'g' },
    { .name = "log-file", .has_arg = true, .flag = nullptr, .val = 'g' },
    { .name = "list", .has_arg = false, .flag = nullptr, .val = 'l' },
    { .name = "help", .has_arg = false, .flag = nullptr, .val = 'h' },
    { .name = nullptr, .has_arg = false, .flag = nullptr, .val = 0 }
};


/* Show program usage text */
static void giveUsage(const char* programName)
{
    fprintf(stderr, 
            "Usage: %s --segment <segment string>...\n"
            "   or: %s --segment <segment string>... --transfer <transfer string>...\n"
            "\nDescription\n"
            "    Benchmark the performance of GPU to GPU RDMA transfer.\n"
            "\nServer arguments\n"
            "  --segment    <segment string>    create a local segment\n"
            "  --export     [export string]     expose a local segment\n"
            "\nClient arguments\n"
            "  --segment    <segment string>    create a local segment\n"
            "  --transfer   <transfer string>   DMA transfer specification\n"
            "\nString format\n"
            "        key1=value1,key2,key3,key4=value4,key5=value5...\n"
            "\nSegment string\n"
            "    size=<size>                    specify size of the segment (required)\n"
            "    a=<no>                         export segment on specified adapter (required for servers)\n"
            "    ls=<id>                        local segment id [default is 0]\n"
            "    gpu=<gpu>                      specify local GPU to host buffer on [omit to host buffer in RAM]\n"
            "\nTransfer string\n"
            "    rn=<id>                        remote node id (required)\n"
            "    rs=<id>                        remote segment id [default is 0]\n"
            "    a=<no>                         local adapter for segment [default is 0]\n"
            "    ls=<id>                        local segment id [default is 0]\n"
            "    pull                           read data from remote buffer instead of writing\n"
            "    ro=<offset>                    offset into remote segment [default is 0]\n"
            "    lo=<offset>                    offset into local segment [default is 0]\n"
            "    size=<size>                    transfer size [default is the size of segment]\n"
            "    repeat=<count>                 number of times to repeat transfer [default is 1]\n"
            "    verify                         calculate checksum of transfer\n"
            "\nOther options\n"
            "  --verbosity      <level>         specify \"error\", \"warn\", \"info\" or \"debug\" log level\n"
            "  --log            <filename>      use a log file instead of stderr for logging\n"
            "  --report         <filename>      use a report file instead of stdout\n"
            "  --list                           show a list of local GPUs and quit\n"
            "  --help                           show this help and quit\n"
            "\n"
            , programName, programName);
}


static void listGpus()
{
    hipError_t err;

    int deviceCount = 0;
    err = hipGetDeviceCount(&deviceCount);
    if (err != hipSuccess)
    {
        throw string(hipGetErrorString(err));
    }

    fprintf(stderr, "\n %2s   %-20s   %-9s   %12s   %7s   %7s   %8s   %6s   %3s   %15s\n",
            "ID", "Device name", "IO addr", "Compute mode", "Managed", "Unified", "Map hmem", "#Async", "L1", "Global mem size");
    fprintf(stderr, "---------------------------------------------------------------------------------------------------------------------\n");

    for (int i = 0; i < deviceCount; ++i)
    {
        hipDeviceProp_t prop;

        err = hipGetDeviceProperties(&prop, i);
        if (err != hipSuccess)
        {
            throw string(hipGetErrorString(err));
        }

        fprintf(stderr, " %2d   %-20s   %02x:%02x.%-3x   %9d.%-2d   %7s   %7s   %8s   %6d   %3s   %10.02f MiB\n",
                i, prop.name, prop.pciBusID, prop.pciDeviceID, prop.pciDomainID,
                prop.major, prop.minor, 
                prop.managedMemory ? "yes" : "no", 
                prop.unifiedAddressing ? "yes" : "no",
                prop.canMapHostMemory ? "yes" : "no",
                prop.asyncEngineCount,
                prop.globalL1CacheSupported ? "yes" : "no",
                prop.totalGlobalMem / (double) (1 << 20)
               );
    }
    fprintf(stderr, "\n");
}


/* Helper function for retrieving key--value pairs from option string */
static const char* nextToken(const char* str, string& key, string& value)
{
    bool readValue = false;

    while (true)
    {
        switch (*str)
        {
            case '=':
                if (readValue)
                {
                    throw string("Invalid string syntax");
                }
                readValue = true;
                break;

            case ',':
                return str + 1;

            case '\0':
                return str;

            default:
                if (readValue)
                {
                    value += *str;
                }
                else
                {
                    key += *str;
                }
                break;
        }

        ++str;
    }
}


/* Helper function for reading a number from a string */
static size_t parseNumber(const string& key, const string& value)
{
    if (value.empty())
    {
        throw string("String key '") + key + string("' expects a numerical value but got nothing");
    }

    size_t idx;
    size_t v = std::stoul(value, &idx, 0);

    if (idx != value.size())
    {
        throw string("String key '") + key + string("' expects a numerical value but got ``") + value + "''";
    }

    return v;
}


static void parseSegmentString(const char* segmentString, SegmentMap& segments)
{
    SegmentPtr segment(new Segment);

    // Parse segment string
    while (*segmentString != '\0')
    {
        string key, value;
        segmentString = nextToken(segmentString, key, value);

        if (key == "size" || key == "s")
        {
            segment->size = parseNumber(key, value);
        }
        else if (key == "local-segment-id" || key == "local-segment" || key == "ls")
        {
            segment->segmentId = parseNumber(key, value);
        }
        else if (key == "adapter" || key == "adapt" || key == "a")
        {
            segment->exports[parseNumber(key, value)] = false;
        }
        else if (key == "device" || key == "gpu")
        {
            segment->deviceId = parseNumber(key, value);
        }
        else if (!key.empty())
        {
            throw string("Unknown string key: ``") + key + "''";
        }
    }

    // Some sanity checking
    if (segment->size == 0)
    {
        throw string("Local segment size must be specified");
    }

    // Check if segment is already specified
    SegmentMap::iterator i = segments.lower_bound(segment->segmentId);
    if (i == segments.end() || segment->segmentId < i->first)
    {
        segments.insert(i, std::make_pair(segment->segmentId, segment));
    }
    else
    {
        throw string("Local segment ") + std::to_string(segment->segmentId) + " was already specified";
    }
}


static void parseTransferString(const char* transferString, TransferVec& transfers)
{
    TransferPtr transfer(new Transfer);
    transfer->repeat = 1;

    // Parse transfer string
    while (*transferString != '\0')
    {
        string key, value;
        transferString = nextToken(transferString, key, value);

        if (key == "remote-node-id" || key == "remote-node" || key == "rn")
        {
            transfer->remoteNodeId = parseNumber(key, value);   
        }
        else if (key == "remote-segment-id" || key == "remote-segment" || key == "rs")
        {
            transfer->remoteSegmentId = parseNumber(key, value);
        }
        else if (key == "adapter" || key == "adapt" || key == "a")
        {
            transfer->localAdapterNo = parseNumber(key, value);
        }
        else if (key == "local-segment-id" || key == "local-segment" || key == "ls")
        {
            transfer->localSegmentId = parseNumber(key, value);
        }
        else if (key == "pull" || key == "read")
        {
            transfer->pull = true;
        }
        else if (key == "remote-offset" || key == "ro")
        {
            transfer->remoteOffset = parseNumber(key, value);
        }
        else if (key == "local-offset" || key == "lo")
        {
            transfer->localOffset = parseNumber(key, value);
        }
        else if (key == "size" || key == "s")
        {
            transfer->size = parseNumber(key, value);
        }
        else if (key == "repeat" || key == "c" || key == "r" || key == "n")
        {
            transfer->repeat = parseNumber(key, value);
        }
        else if (key == "verify")
        {
            //transfer.verify = true;
        }
    }


    // Some sanity checking
    if (transfer->remoteNodeId == 0)
    {
        throw string("Remote node id must be specified for transfers");
    }

    if (transfer->repeat == 0)
    {
        throw string("Transfers can not be repeated 0 times");
    }

    transfers.push_back(transfer);
}


static Log::Level parseVerbosity(const char* argument, uint level)
{
    if (argument == nullptr)
    {
        // No argument given, increase verbosity level
        return level < Log::Level::DEBUG ? (Log::Level) (level + 1) : Log::Level::DEBUG;
    }
    else if (strcmp(argument, "error") == 0)
    {
        return Log::Level::ERROR;
    }
    else if (strcmp(argument, "warn") == 0)
    {
        return Log::Level::WARN;
    }
    else if (strcmp(argument, "info") == 0)
    {
        return Log::Level::INFO;
    }
    else if (strcmp(argument, "debug") == 0)
    {
        return Log::Level::DEBUG;
    }

    // Try to parse verbosity level as a number
    char* ptr = nullptr;
    level = strtoul(optarg, &ptr, 10);

    if (ptr == nullptr || *ptr != '\0')
    {
        throw "Unknown log level: ``" + string(optarg) + "''";
    }

    return level < Log::Level::DEBUG ? (Log::Level) (level + 1) : Log::Level::DEBUG;
}


/* Parse command line options */
static void parseArguments(int argc, char** argv, SegmentMap& segments, TransferVec& transfers, Log::Level& logLevel, string& logFile, string& reportFile)
{
    int option;
    int index;

    while ((option = getopt_long(argc, argv, "-:s:t:f:g:vlh", options, &index)) != -1)
    {
        switch (option)
        {
            case ':': // Missing value for option
                fprintf(stderr, "Argument %s requires a value\n", argv[optind - 1]);
                giveUsage(argv[0]);
                throw 1;

            case '?': // Unknown option
                fprintf(stderr, "Unknown option: ``%s''\n", argv[optind - 1]);
                giveUsage(argv[0]);
                throw 1;

            case 'h': // Show help
                giveUsage(argv[0]);
                throw 0;

            case 'l': // List GPUs
                listGpus();
                throw 0;

            case 'v': // Increase verbosity level
                logLevel = parseVerbosity(optarg, logLevel);
                break;

            case 'f': // Set report file
                reportFile = optarg;
                break;

            case 'g': // Set log file
                logFile = optarg;
                break;

            case 's': // Parse local segment options
                parseSegmentString(optarg, segments);
                break;

            case 't': // Parse transfer string
                parseTransferString(optarg, transfers);
                break;
        }
    }
}


int main(int argc, char** argv)
{
    SegmentMap segments;
    TransferVec transfers;
    FILE* logFile = stderr;
    FILE* reportFile = stdout;
    string logFilename, reportFilename;
    Log::Level logLevel = Log::Level::ERROR;

    // Parse command line arguments
    try
    {
        parseArguments(argc, argv, segments, transfers, logLevel, logFilename, reportFilename);
    }
    catch (int error)
    {
        return error;
    }
    catch (const string& error)
    {
        fprintf(stderr, "%s\n", error.c_str());
        return 1;
    }

    // Do some sanity checking
    if (segments.empty())
    {
        fprintf(stderr, "No segments specified\n");
        return 1;
    }
    else if (transfers.empty())
    {
        for (SegmentMap::const_iterator it = segments.begin(); it != segments.end(); ++it)
        {
            if (it->second->exports.empty())
            {
                fprintf(stderr, "Server mode specified, but no segment %u has no exports\n", it->first);
                return 1;
            }
        }
    }

    // Initialize SISCI API
    sci_error_t err = SCI_ERR_OK;
    SCIInitialize(0, &err);
    if (err != SCI_ERR_OK)
    {
        fprintf(stderr, "Failed to initialize SISCI API\n");
        return 2;
    }

    // Open log file
    if (!logFilename.empty())
    {
        logFile = fopen(logFilename.c_str(), "a");
        if (logFile == nullptr)
        {
            SCITerminate();
            fprintf(stderr, "Failed to open log file: %s\n", strerror(errno));
            return 1;
        }
    }
    Log::init(logFile, logLevel);
    Log::info("New run started...");

    if (transfers.empty())
    {
        // Catch ctrl+c from terminal
        auto stopServer = [](int) {
            stopBenchmarkServer();
        };

        signal(SIGTERM, (sig_t) stopServer);
        signal(SIGINT, (sig_t) stopServer);

        // No transfers specified, run as server
        if (runBenchmarkServer(segments) != 0)
        {
            fprintf(stderr, "SERVER FAILED\n");
        }
    }
    else
    {
        // Open report file
        if (!reportFilename.empty())
        {
            reportFile = fopen(reportFilename.c_str(), "a");
            if (reportFile == nullptr)
            {
                fprintf(stderr, "Failed to open report file: %s\n", strerror(errno));
                SCITerminate();
                fclose(logFile);
                return 1;
            }
        }

        // Run benchmark client
        if (runBenchmarkClient(segments, transfers) != 0)
        {
            fprintf(stderr, "CLIENT FAILED\n");
        }

        fflush(reportFile);
        fclose(reportFile);
    }

    // Destroy any active SISCI handles
    transfers.clear();
    segments.clear();

    // Terminate SISCI API
    SCITerminate();
    fclose(logFile);

    return 0;
}
